#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

#define CUDA_CHECK_RETURN(value) { \
hipError_t _m_cudaStat = value; \
if (_m_cudaStat != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1); \
} }

#define BLOCK_SIZE_LIMIT (1024)
#define GRID_SIZE_LIMIT (65535)

__global__ void multiply(int N, int dsize, int *M, int *D, int* T, int *R) {
    for (int rindex = blockIdx.x * blockDim.x + threadIdx.x; rindex < N;
	    rindex += gridDim.x * blockDim.x) {
	for(int i = 0; i < dsize; i++) {
	    int tindex = D[i] + rindex;
	    if(tindex>=0 && tindex < N)
		R[rindex] += *(M + rindex * dsize + i) * T[tindex];
	}
    }
}

void clean(int *T, int n) {
    for(int i = 0; i < n; i++)
	T[i] = 0;
}

void print_array(int *T, int n) {
    for(int i = 0; i < n; i++)
	printf("%d\n", T[i]);
}

int main() {
    int n;
    scanf("%d", &n);

    // length of the vector describing input matrix
    int dsize;
    scanf("%d", &dsize);

    if(n==0)
	return 0;

    int block_size = BLOCK_SIZE_LIMIT;
    int threads = min(n, GRID_SIZE_LIMIT * block_size);
    int grid_size = (threads+block_size-1)/block_size;

    // Matrix
    int *mat;
    int *devMat;
    hipHostMalloc((void**) &mat, n * dsize * sizeof(int));
    hipMalloc((void**) &devMat, n * dsize * sizeof(int));

    // Vector describing matrix
    int *D;
    int *devD;
    hipHostMalloc((void**) &D, dsize * sizeof(int));
    hipMalloc((void**) &devD, dsize * sizeof(int));

    // Vector we multiply by (Times)
    int *T;
    int *devT;
    hipHostMalloc((void**) &T, n * sizeof(int));
    hipMalloc((void**) &devT, n * sizeof(int));

    // result
    int *R;
    int *devR;
    hipHostMalloc((void**) &R, n * sizeof(int));
    hipMalloc((void**) &devR, n * sizeof(int));
    clean(R, n);

    for(int i = 0; i < dsize; i++) {
	scanf("%d", &D[i]);
	for(int j = 0; j < n; j++)
	    scanf("%d", mat+j*dsize + i);
	// n rows, dsize columns
	// each column is some diagonal of original matrix
    }

    for(int i = 0; i < n; i++)
	scanf("%d", &T[i]);

    hipMemcpy(devMat, mat, sizeof(int) * n *dsize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, sizeof(int) * dsize, hipMemcpyHostToDevice);
    hipMemcpy(devT, T, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(devR, R, sizeof(int) * n, hipMemcpyHostToDevice);

    multiply<<<grid_size, block_size>>>(n, dsize, devMat, devD, devT, devR);

    CUDA_CHECK_RETURN(hipGetLastError());

    hipMemcpy(R, devR, sizeof(int) * n, hipMemcpyDeviceToHost);
    print_array(R, n);

    hipHostFree((void*) mat);
    hipFree((void*) devMat);
    hipHostFree((void*) D);
    hipFree((void*) devD);
    hipHostFree((void*) T);
    hipFree((void*) devT);
    hipHostFree((void*) R);
    hipFree((void*) devR);

    return 0;
}
